
#include <hip/hip_runtime.h>
#include <stdio.h>

long reduce_gold(long *data, int len) {
   long res = 0;
   for (long i = 0; i < len; i++) {
      res += data[i];
   }
   return res;
}

__global__ void reduce1(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[i];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      if (tid % (2*s) == 0) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[i];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      int index = 2 * s * tid;
      if (index < blockDim.x) {
       sdata[index] += sdata[index + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce3(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[i];
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce4(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * 2 * blockDim.x;
   sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__device__ void warpReduce(volatile int *sdata, int tid) {
   sdata[tid] += sdata[tid + 32];
   sdata[tid] += sdata[tid + 16];
   sdata[tid] += sdata[tid + 8];
   sdata[tid] += sdata[tid + 4];
   sdata[tid] += sdata[tid + 2];
   sdata[tid] += sdata[tid + 1];
}

__global__ void reduce5(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x*2 * blockDim.x;
   sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 32; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }

   if (tid < 32) warpReduce(sdata, tid);
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize>
__device__ void warpReduceT(volatile int* sdata, int tid) {
   if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
   if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
   if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
   if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
   if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
   if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}


template <unsigned int blockSize>
__global__ void reduce6(long *g_idata, long *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x*2;
   sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
   __syncthreads();

   if(blockSize >= 1024){
        if(tid < 512){
            sdata[tid] += sdata[tid+512];
        }
        __syncthreads();
    }
    if(blockSize >= 512){
        if(tid < 256){
            sdata[tid] += sdata[tid+256];
        }
        __syncthreads();
    }
    if(blockSize >= 256){
        if(tid < 128){
            sdata[tid] += sdata[tid+128];
        }
        __syncthreads();
    }
    if(blockSize >= 128){
        if(tid < 64){
            sdata[tid] += sdata[tid+64];
        }
        __syncthreads();
    }

   if (tid < 32) warpReduceT<blockSize>(sdata, tid);
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

void kernel6 (long *d_idata, long *d_odata, int num_thread, int num_block) {
   switch (num_thread) {
      case 1024:
         reduce6<1024><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 512:
         reduce6<512><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 256:
         reduce6<256><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 128:
         reduce6<128><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 64:
         reduce6<64><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 32:
         reduce6<32><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 16:
         reduce6<16><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 8:
         reduce6<8><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 4:
         reduce6<4><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 2:
         reduce6<2><<<num_block, num_thread>>>(d_idata, d_odata); break;
      case 1:
         reduce6<1><<<num_block, num_thread>>>(d_idata, d_odata); break;
   }
}

__global__ void reduce7(long *d_idata, long *d_odata, int num_element) {
   extern __shared__ int sdata[];
   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
   unsigned int gridSize = blockDim.x * 2 * gridDim.x;

   sdata[tid] = 0;
   while (i < num_element) {
      sdata[tid] += d_idata[i] + d_idata[i + blockDim.x];
      i += gridSize;
   }
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 32; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }

   if (tid < 32) warpReduce(sdata, tid);

   if (tid == 0) d_odata[blockIdx.x] = sdata[0];
}

int main() {
   int num_element = 1 << 22;
   int num_thread = 256;
   int num_block = num_element / num_thread;

   long *h_idata, *d_idata, *h_odata, *d_odata;

   h_idata = (long *)malloc(sizeof(long) * num_element);
   h_odata = (long *)malloc(sizeof(long) * num_block);

   hipMalloc((void **)&d_idata, sizeof(long)*num_element);
   hipMalloc((void **)&d_odata, sizeof(long)*num_block);

   for (long i = 0; i < num_element; i++) {
      h_idata[i] = i;
   }

   long sum_gpu = 0, sum = 0;

   float milli;
   hipEvent_t start, end;
   hipEventCreate(&start);
   hipEventCreate(&end);

   printf("4M elements for reduction\n\n");

   hipEventRecord(start);
   sum = reduce_gold(h_idata, num_element);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("CPU: Elapsed time = %.4f ms\n\n", milli);

   // cudaEventRecord(start);
   hipMemcpy(d_idata, h_idata, sizeof(long)*num_element, hipMemcpyHostToDevice);
   // cudaEventRecord(end);
   // cudaEventSynchronize(end);
   // cudaEventElapsedTime(&milli, start, end);
   // printf("Bandwidth: %.2f GB/s\n", sizeof(long)*num_element*1.0 / (milli * 1000000));

   hipEventRecord(start);
   reduce1<<<num_block, num_thread>>>(d_idata, d_odata);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction1: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   reduce2<<<num_block, num_thread>>>(d_idata, d_odata);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction2: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   reduce3<<<num_block, num_thread>>>(d_idata, d_odata);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction3: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   reduce4<<<num_block, num_thread>>>(d_idata, d_odata);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction4: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   reduce5<<<num_block, num_thread>>>(d_idata, d_odata);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction5: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   kernel6(d_idata, d_odata, num_thread, num_block);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction6: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   hipEventRecord(start);
   reduce7<<<num_block, num_thread>>>(d_idata, d_odata, num_element);
   hipEventRecord(end);
   hipEventSynchronize(end);
   hipEventElapsedTime(&milli, start, end);
   printf("Reduction7: Elapsed time = %.4f ms", milli);
   hipMemcpy(h_odata, d_odata, sizeof(long)*num_block, hipMemcpyDeviceToHost);
   for (unsigned int i = 0; i < num_block; i++) {
      sum_gpu += h_odata[i];
   }
   printf(sum_gpu-sum == 0 ? "   correct\n\n" : "   error\n\n");
   sum_gpu = 0;

   free(h_idata);
   free(h_odata);
   hipFree(d_idata);
   hipFree(d_odata);

   return 0;
}