
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reduce1(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[tid];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      if (tid % (2*s) == 0) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[tid];
   __syncthreads();

   for (unsigned int s = 1; s < blockDim.x; s *= 2) {
      int index = 2 * s * tid;
      if (index < blockDim.x) {
       sdata[index] += sdata[index + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce3(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
   sdata[tid] = g_idata[tid];
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce4(int *g_idata, int *g_odata) {
   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = threadIdx.x + blockIdx.x*2 * blockDim.x;
   sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
   __syncthreads();

   for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
      if (tid < s) {
       sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
   }
   
   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {

}